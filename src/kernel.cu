#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

#define CELL_WIDTH_FACTOR 2.0f

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  //initialize numObjects here
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = CELL_WIDTH_FACTOR * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
  
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  //these two arrays only use to store cell information, so no need to be as many as numObjects
  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    glm::vec3 perceived_center(0.f,0.f,0.f);
    glm::vec3 avoidance_velocity(0.f, 0.f, 0.f);
    glm::vec3 perceived_velocity(0.f, 0.f, 0.f);
    glm::vec3 return_vel(0.f, 0.f, 0.f);
    float neighbor_count_rule1 = 0;
    float neighbor_count_rule3 = 0;

    //pre load all needed data
    glm::vec3 curr_boid_pos = pos[iSelf];
    for (int idx = 0; idx < N; ++idx)
    {
        //if b = boid skip the rest actions
        if (idx == iSelf) continue;
        
        //load current boid pos
        glm::vec3 idx_boid_pos = pos[idx];
        float dist = glm::distance(idx_boid_pos, curr_boid_pos);
        // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
        if (dist < rule1Distance)
        {
            perceived_center += idx_boid_pos;
            neighbor_count_rule1++;
        }
        
        // Rule 2: boids try to stay a distance d away from each other
        if (dist < rule2Distance)
        {
            avoidance_velocity -= (idx_boid_pos - curr_boid_pos);
        }

        // Rule 3: boids try to match the speed of surrounding boids
        if (dist < rule3Distance)
        {
            perceived_velocity += vel[idx];
            neighbor_count_rule3++;
        }
    }

    //if we use N-1, the particles will shrink to the center of cube  -- helped by Hannar
    glm::vec3 rule1_component = glm::vec3(0.f, 0.f, 0.f);
    glm::vec3 rule2_component = glm::vec3(0.f, 0.f, 0.f);
    glm::vec3 rule3_component = glm::vec3(0.f, 0.f, 0.f);
    if (neighbor_count_rule1 > 0)
    {
        rule1_component = (perceived_center / neighbor_count_rule1 - curr_boid_pos) * rule1Scale;
    }
    rule2_component = avoidance_velocity * rule2Scale;
    if (neighbor_count_rule3 > 0)
    {
        rule3_component = (perceived_velocity / neighbor_count_rule3) * rule3Scale;
    }

    //helped by Hanna ReadMe Rule part sum all rules' and current velocity
    return_vel += vel[iSelf] + rule1_component + rule2_component + rule3_component;
  
    return return_vel;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
    //Compute the index of current thread
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index > N)
    {
        return;
    }
    glm::vec3 new_velocity = computeVelocityChange(N, index, pos, vel1);
  // Clamp the speed
    float curr_speed = glm::length(new_velocity);
    //if the total speed of vel is larger than maxSpeed, we normalize the vel and apply the maxSpeed we allow  -- do we need to care negative speed?
    if (curr_speed > maxSpeed)
    {
        new_velocity = glm::normalize(new_velocity) * maxSpeed;
    }
    
  // Record the new velocity into vel2. Question: why NOT vel1? --- because other boids might need that
    vel2[index] = new_velocity;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    //compute the correspond index in x, y, z axis and use gridIndex3Dto1D to store the actual 1D index  -- why we need inverseCellWidth  -- mult is faster than divide
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    if (idx >= N) {
        return;
    }
        //pre store info
        glm::vec3 curr_pos = pos[idx];
        int idx_x = (curr_pos.x - gridMin.x) * inverseCellWidth;
        int idx_y = (curr_pos.y - gridMin.y) * inverseCellWidth;
        int idx_z = (curr_pos.z - gridMin.z) * inverseCellWidth;

        //combine to get the 1D index
        int gridIndex = gridIndex3Dto1D(idx_x, idx_y, idx_z, gridResolution);

        //store to indices and gridIndices correspondingly
        // - Set up a parallel array of integer indices as pointers to the actual
        //   boid data in pos and vel1/vel2
        indices[idx] = idx;
        gridIndices[idx] = gridIndex;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
    //what happen for those we don't include any particle? is there an identifier to show its identity?

    //so why do we need th Particles array?  we don't even pass in it -- the arranged start and end indices are for us to access dev_particleArrayIndices, which has been sorted
    //may have better way to do
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    if (idx >= N) {
        return;
    }
    int target_grid_index = particleGridIndices[idx];
    //look at the one before and the one after, if diff, store
    //head must be a start
    if (idx == 0)
    {
        gridCellStartIndices[target_grid_index] = idx;
    }

    if (idx == N - 1)
    {
        gridCellEndIndices[target_grid_index] = idx;
        return;
    }


    //check one before and one after
    if (target_grid_index != 0 && target_grid_index != particleGridIndices[idx - 1])
    {
        //start of a cell
        gridCellStartIndices[target_grid_index] = idx;
    
    }

    if (target_grid_index != N - 1 && target_grid_index != particleGridIndices[idx + 1])
    {
        gridCellEndIndices[target_grid_index] = idx;
        
    }

    //int next_grid_index = particleGridIndices[idx + 1];
    //if (target_grid_index != next_grid_index)
    //{
    //    gridCellStartIndices[next_grid_index] = idx + 1;
    //    gridCellEndIndices[target_grid_index] = idx;
    //}

}

//very similar to photon mapping
__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    if (idx >= N) {
        return;
    }
    glm::vec3 curr_boid_pos = pos[idx];
    int idx_x = (curr_boid_pos.x - gridMin.x) * inverseCellWidth;
    int idx_y = (curr_boid_pos.y - gridMin.y) * inverseCellWidth;
    int idx_z = (curr_boid_pos.z - gridMin.z) * inverseCellWidth;

    //combine to get the 1D index
    int gridIndex = gridIndex3Dto1D(idx_x, idx_y, idx_z, gridResolution);
  // - Identify which cells may contain neighbors. This isn't always 8.
    //by calculating those cells that interact with the sphere with the neighbor_radius(std::max(std::max(rule1Distance, rule2Distance), rule3Distance))  -not allowed
    //float neighbor_radius = cellWidth / CELL_WIDTH_FACTOR;
    const float neighbor_radius = glm::max(glm::max(rule1Distance, rule2Distance), rule3Distance);
    int max_x = (curr_boid_pos.x + neighbor_radius - gridMin.x) * inverseCellWidth;
    max_x = max_x > gridResolution - 1 ? gridResolution - 1 : max_x;
    int min_x = (curr_boid_pos.x - neighbor_radius - gridMin.x) * inverseCellWidth;
    min_x = min_x < 0 ? 0 : min_x;
    int max_y = (curr_boid_pos.y + neighbor_radius - gridMin.y) * inverseCellWidth;
    max_y = max_y > gridResolution - 1 ? gridResolution - 1 : max_y;
    int min_y = (curr_boid_pos.y + neighbor_radius - gridMin.y) * inverseCellWidth;
    min_y = min_y < 0 ? 0 : min_y;
    int max_z = (curr_boid_pos.z + neighbor_radius - gridMin.z) * inverseCellWidth;
    max_z = max_z > gridResolution - 1 ? gridResolution - 1 : max_z;
    int min_z = (curr_boid_pos.z + neighbor_radius - gridMin.z) * inverseCellWidth;
    min_z = min_z < 0 ? 0 : min_z;

  // - For each cell, read the start/end indices in the boid pointer array.
    glm::vec3 perceived_center(0.f, 0.f, 0.f);
    glm::vec3 avoidance_velocity(0.f, 0.f, 0.f);
    glm::vec3 perceived_velocity(0.f, 0.f, 0.f);
    glm::vec3 new_velocity(0.f, 0.f, 0.f);
    float neighbor_count_rule1 = 0;
    float neighbor_count_rule3 = 0;
    for (int x_cord_idx = min_x; x_cord_idx <= max_x; ++x_cord_idx)
    {
        for (int y_cord_idx = min_y; y_cord_idx <= max_y; ++y_cord_idx)
        {
            for (int z_cord_idx = min_z; z_cord_idx <= max_z; ++z_cord_idx)
            {
                int curr_gridIndex = gridIndex3Dto1D(x_cord_idx, y_cord_idx, z_cord_idx, gridResolution);
                //read the start/end indices
                int start = gridCellStartIndices[curr_gridIndex];
                int end = gridCellEndIndices[curr_gridIndex];
                if (start == -1 || end == -1) {
                    continue; //no boid in this cell
                }
                else
                {
                    // - Access each boid in the cell and compute velocity change from
                    //   the boids rules, if this boid is within the neighborhood distance.
                    for (int boid_array_idx = start; boid_array_idx <= end; ++boid_array_idx)
                    {
                        //the boid_array_idx is only the index in particleArray, need to load it out -- buggy for only few particles moving
                        int boid_idx = particleArrayIndices[boid_array_idx];

                        if (boid_idx == idx) continue;

                        //pre load temp boid pos
                        glm::vec3 idx_boid_pos = pos[boid_idx];
                        float dist = glm::distance(idx_boid_pos, curr_boid_pos);
                        // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
                        if (dist < rule1Distance)
                        {
                            perceived_center += idx_boid_pos;
                            neighbor_count_rule1++;
                        }

                        // Rule 2: boids try to stay a distance d away from each other
                        if (dist < rule2Distance)
                        {
                            avoidance_velocity -= (idx_boid_pos - curr_boid_pos);
                        }

                        // Rule 3: boids try to match the speed of surrounding boids
                        if (dist < rule3Distance)
                        {
                            perceived_velocity += vel1[idx];
                            neighbor_count_rule3++;
                        }
                        
                    }
                
                }
            }
        
        }
    
    }

    //compute the new velocity
    glm::vec3 rule1_component = glm::vec3(0.f, 0.f, 0.f);
    glm::vec3 rule2_component = glm::vec3(0.f, 0.f, 0.f);
    glm::vec3 rule3_component = glm::vec3(0.f, 0.f, 0.f);
    if (neighbor_count_rule1 > 0)
    {
        rule1_component = (perceived_center / neighbor_count_rule1 - curr_boid_pos) * rule1Scale;
    }
    rule2_component = avoidance_velocity * rule2Scale;
    if (neighbor_count_rule3 > 0)
    {
        rule3_component = (perceived_velocity / neighbor_count_rule3) * rule3Scale;
    }
    new_velocity += vel1[idx] + rule1_component + rule2_component + rule3_component;
  // - Clamp the speed change before putting the new speed in vel2
    float curr_speed = glm::length(new_velocity);
    //if the total speed of vel is larger than maxSpeed, we normalize the vel and apply the maxSpeed we allow
    if (curr_speed > maxSpeed)
    {
        new_velocity = glm::normalize(new_velocity) * maxSpeed;
    }

    vel2[idx] = new_velocity;
}



__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
    int gridSize = (numObjects + blockSize - 1) / blockSize; //helped by Gangzheng Tong
    dim3 blocksPerGrid(gridSize);
    //first compute the new velocity
    kernUpdateVelocityBruteForce <<< blocksPerGrid, threadsPerBlock >>> (numObjects, dev_pos, dev_vel1, dev_vel2);

    //Then update the pos
    kernUpdatePos << < blocksPerGrid, threadsPerBlock >> > (numObjects, dt, dev_pos, dev_vel2);

  // TODO-1.2 ping-pong the velocity buffers -- swap content
    hipMemcpy(dev_vel1, dev_vel2, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:

  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.

    //set up blocks
    int gridSizeParticle = (numObjects + blockSize - 1) / blockSize; //helped by Gangzheng Tong
    dim3 blocksPerGridParticle(gridSizeParticle);
    int gridSizeGridCell = (gridCellCount + blockSize - 1) / blockSize;
    dim3 blocksPerGridGridCell(gridSizeGridCell);
    //call kernel
    kernComputeIndices << < blocksPerGridParticle, threadsPerBlock >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices); //sort by grid index

  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
    //first initialize the two indices list to be -1
    kernResetIntBuffer <<< blocksPerGridGridCell, threadsPerBlock >>> (gridCellCount, dev_gridCellStartIndices, -1);
    kernResetIntBuffer <<< blocksPerGridGridCell, threadsPerBlock >>> (gridCellCount, dev_gridCellEndIndices, -1);

    //then call the kernel to compute those who contain boids
    kernIdentifyCellStartEnd <<< blocksPerGridParticle, threadsPerBlock >>> (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
  // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchScattered << < blocksPerGridParticle, threadsPerBlock >> > (
        numObjects, gridSideCount, gridMinimum,
        gridInverseCellWidth, gridCellWidth,
        dev_gridCellStartIndices, dev_gridCellEndIndices,
        dev_particleArrayIndices,
        dev_pos, dev_vel1, dev_vel2);
  // - Update positions
    kernUpdatePos << < blocksPerGridParticle, threadsPerBlock >> > (numObjects, dt, dev_pos, dev_vel2);
  // - Ping-pong buffers as needed
    hipMemcpy(dev_vel1, dev_vel2, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  //2.1 free
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
}

void Boids::unitTest() {
    // LOOK-1.2 Feel free to write additional tests here.

    // test unstable sort
    int *dev_intKeys;
    int *dev_intValues;
    int N = 10;

    std::unique_ptr<int[]>intKeys{ new int[N] };
    std::unique_ptr<int[]>intValues{ new int[N] };

    intKeys[0] = 0; intValues[0] = 0;
    intKeys[1] = 1; intValues[1] = 1;
    intKeys[2] = 0; intValues[2] = 2;
    intKeys[3] = 3; intValues[3] = 3;
    intKeys[4] = 0; intValues[4] = 4;
    intKeys[5] = 2; intValues[5] = 5;
    intKeys[6] = 2; intValues[6] = 6;
    intKeys[7] = 0; intValues[7] = 7;
    intKeys[8] = 5; intValues[8] = 8;
    intKeys[9] = 6; intValues[9] = 9;

    hipMalloc((void**)&dev_intKeys, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

    hipMalloc((void**)&dev_intValues, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

    dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

    std::cout << "before unstable sort: " << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << "  key: " << intKeys[i];
        std::cout << " value: " << intValues[i] << std::endl;
    }

    // How to copy data to the GPU
    hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

    // Wrap device vectors in thrust iterators for use with thrust.
    thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
    thrust::device_ptr<int> dev_thrust_values(dev_intValues);
    // LOOK-2.1 Example for using thrust::sort_by_key
    thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

    // How to copy data back to the CPU side from the GPU
    hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
    checkCUDAErrorWithLine("memcpy back failed!");

    std::cout << "after unstable sort: " << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << "  key: " << intKeys[i];
        std::cout << " value: " << intValues[i] << std::endl;
    }

    // cleanup
    hipFree(dev_intKeys);
    hipFree(dev_intValues);
    checkCUDAErrorWithLine("hipFree failed!");
    return;
}